#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <string>
#include <algorithm> 
#include <mmio_cust.h>
#include <common_defs.h>
#include <numeric>
#include <data_struct.h>
#include <parse_mat.h>
#include <csr2tile.h>
#include <tile_spgemm.h>

int main(){


    SMatrix *matrixA = (SMatrix *)malloc(sizeof(SMatrix));
	SMatrix *matrixB = (SMatrix *)malloc(sizeof(SMatrix));

    const char * filename = "pwtk.mtx";
    
    parse_mtx(&matrixA->m, &matrixA->n, &matrixA->nnz, &matrixA->isSymmetric, &matrixA->rowpointer, &matrixA->columnindex, &matrixA->value, filename);
    
    // Parse MTX A
    std::cout << "Parsed  MAT A :m = " << matrixA->m << ", " << "n = " << matrixA->n << "\n";

    // Parse MTX B
    parse_mtx(&matrixB->m, &matrixB->n, &matrixB->nnz, &matrixB->isSymmetric, &matrixB->rowpointer, &matrixB->columnindex, &matrixB->value, filename);
    std::cout << "Parsed  MAT B :m = " << matrixB->m << ", " << "n = " << matrixB->n << "\n";

    // Convert A to Tile row major format?
    csr2tile_row_major(matrixA);

    // Convert B to Tile column major format
    csr2tile_col_major(matrixB);

    int blk_intersec_bitmask_len = ceil((double)matrixA->tilen / 32.0);
    double densityA = (double)matrixA->numtile / ((double)matrixA->tilem*(double)matrixA->tilen);
    double densityB = (double)matrixB->numtile / ((double)matrixB->tilem*(double)matrixB->tilen);


    long long int lengthA = (long long int) (matrixA->tilem) * (long long int)( blk_intersec_bitmask_len) ;

    unsigned int *blk_intersec_bitmask_A = (unsigned int *)malloc(lengthA* sizeof(unsigned int));
    memset(blk_intersec_bitmask_A, 0, lengthA * sizeof(unsigned int));
    for (int i = 0; i < matrixA->tilem; i++)
    {
        for (int j = matrixA->tile_ptr[i]; j < matrixA->tile_ptr[i + 1]; j++)
        {
            int idx = matrixA->tile_columnidx[j];
            unsigned int bitmask = 1;
            bitmask <<=  (31- (idx % 32));
            long long int pos = (long long int)i * (long long int)blk_intersec_bitmask_len + idx / 32;
            blk_intersec_bitmask_A[pos] |= bitmask;
        }
    }

    long long int lengthB = (long long int) (matrixB->tilen) * (long long int)(blk_intersec_bitmask_len) ;

    unsigned int *blk_intersec_bitmask_B = (unsigned int *)malloc(lengthB * sizeof(unsigned int));
    memset(blk_intersec_bitmask_B, 0, lengthB * sizeof(unsigned int));
    for (int i = 0; i < matrixB->tilen; i++)
    {
        for (int j = matrixB->csc_tile_ptr[i]; j < matrixB->csc_tile_ptr[i+1]; j++)
        {
            int idx = matrixB->csc_tile_rowidx[j];
            unsigned int bitmask = 0x1;
            bitmask <<= (31 - (idx % 32));
            long long int pos = (long long int)i * (long long int )blk_intersec_bitmask_len + idx / 32;
            blk_intersec_bitmask_B[pos] |= bitmask;
        }
    }


    // generate rowidx of blockA
    int *tile_rowidx_A = (int *)malloc (matrixA->numtile * sizeof(int ) );
    for (int i = 0; i < matrixA->tilem; i++)
    {
        for (int j = matrixA->tile_ptr[i]; j < matrixA->tile_ptr[i+1]; j++)
        {
            tile_rowidx_A[j] = i;
        }
    }

    // --------------------------------------------------------------------------------------------------------
    SMatrix *matrixC = (SMatrix *)malloc(sizeof(SMatrix));
    
    unsigned long long int nnzC_computed;
    double compression_rate = 0;
    double time_tile = 0;
    double gflops_tile = 0;
    double time_step1 =0,time_step2 =0,time_step3 =0,time_malloc=0; 


}